#include "hip/hip_runtime.h"
/*******************************************************************************
* File Name: main.cpp
* Author: Mathurin Champemont
* Created Date: 2023-06-14
* Last Modified: 2023-06-14
* Description: main file of the crowd simulation with parallelization on GPU. Contains only the main program.
*******************************************************************************/

// Include necessary libraries here
#include "kernel.hpp"

int main(int argc, char const *argv[])
{
    simParam _simParam;
    settings _settings;
    kernelParam _kernelParam;

    if( _settings.print > 2 )std::cout  << " ### Init simulation ###" << std::endl;
    srand(time(NULL));
    initSimSettings(argc, argv, &_simParam, &_settings);
    initPopulationPositionMap(&_simParam, _settings);
    initKernelParam(&_kernelParam, _simParam, _settings);
    printMap(_simParam, _settings);

    while (_simParam.isFinish == 0){
        if (_simParam.pInSim == 0) _simParam.isFinish = 1; 
        
        //progressBar(simDimP - simPIn, simDimP, 100, 0);
        shuffleIndex(&_simParam, _settings);
        
        // MODEL
        switch (_settings.model){
            case 0: // MODEL : sage ignorant
                // TO DO
                _simParam.pInSim --;

                // kernel_model1_GPU<<<blocks,threads>>>(dev_populationPosition, dev_map, dev_simPIn, cost, simExit, simDim, simDimP);
                
                for (size_t tid = 0; tid < _simParam.nbIndividual; tid++)
                {
                    // position de l'individue tid
                    uint2 pos    = make_uint2(_simParam.populationPosition[tid].x, _simParam.populationPosition[tid].y);
                    uint2 delta  = make_uint2(_simParam.exit.x - pos.x, _simParam.exit.y-pos.y);
                    uint  maxDim = max(abs(delta.x), abs(delta.y));
                    uint2 move   = make_uint2(delta.x / maxDim, delta.y / maxDim);
                    std::cout <<"c "<<pos.x<<" "<<pos.y<<"\te "<<_simParam.exit.x<<" "<<_simParam.exit.y<<"\td "<<delta.x<<" "<<delta.y<<"\tm "<<move.x<<" "<<move.y;

                    // on regarde si la case est disponible 
                    if(_simParam.map[ _simParam.dimension.x * (pos.y+move.y) + (pos.x + move.x)] == -1){ // if is EMPTY
                        std::cout <<"-> moove" << std::endl;
                        _simParam.populationPosition[tid] = make_uint2(pos.x + move.x, pos.y + move.y);
                        // Temporaire
                        _simParam.map[_simParam.dimension.x * pos.y + pos.x]                        = -1;
                        _simParam.map[_simParam.dimension.x * (pos.y+move.y) + (pos.x + move.x)]    = tid;
                    }
                    else std::cout << std::endl;
                }
                break;

            case 1: // MDOEL : Impatient ignorant
            case 2: // MDOEL : Forcée
            case 3: // MDOEL : Conne de vision
            case 4: // MDOEL : Meilleur coût
            case 5: // MDOEL : Meilleur déplacement 
            default:
                _simParam.pInSim--;
                break;
        }
        // printMap(_simParam, _settings);
        // EXPORT 
        switch (_settings.exportType){
            case 1:
                // TO DO  
                break;
            
            default:
                break;
        }
    }
    
    if( _settings.print > 2 )std::cout << " \t> Cuda Copy ";
    //hipMemcpy(outMove           , dev_outMove           , sizeof(uint)                      , hipMemcpyDeviceToHost);
    //hipMemcpy(populationPosition, dev_populationPosition, 2 * sizeof(uint) * simDimP        , hipMemcpyDeviceToHost);
    //hipMemcpy(map               , dev_map               , simDim.x * simDim.y * sizeof(uint ) , hipMemcpyDeviceToHost);
    if( _settings.print > 2 )std::cout  << " OK " << std::endl;

    std::cout << std::endl;
    printMap(_simParam, _settings);

    return 0;
}
