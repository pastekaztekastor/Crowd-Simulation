#include "hip/hip_runtime.h"
/*******************************************************************************
* File Name: kernel.cu
* Author: Mathurin Champemont
* Created Date: 2023-06-14
* Last Modified: 2023-06-14
* Description: 
*******************************************************************************/

#include "kernel.hpp"

void model1_GPU( 
    unsigned int ***    populationPosition, 
    enum _Element ***   map, 
    unsigned int *      simPIn, 
    unsigned int **     cost, 
    unsigned int *      simExit,  
    unsigned int        simDimX,  
    unsigned int        simDimY,  
    unsigned int        simDimP, 
    unsigned int        settings_print 
){

    unsigned int ***    dev_populationPosition;
    enum _Element ***   dev_map;
    unsigned int *      dev_simPIn;
    unsigned int **     dev_cost;
    unsigned int *      dev_simDimP, 
    unsigned int *      dev_simExit;
    unsigned int *      dev_outMove;
    //unsigned int        outMove = 0;

    hipMalloc((void**) &dev_populationPosition , sizeof(unsigned int *** ));
    hipMalloc((void**) &dev_map                , sizeof(enum _Element ***));
    hipMalloc((void**) &dev_simPIn             , sizeof(unsigned int *   ));
    hipMalloc((void**) &dev_cost               , sizeof(unsigned int **  ));
    hipMalloc((void**) &dev_simExit            , sizeof(unsigned int *   ));
    hipMalloc((void**) &dev_outMove            , sizeof(unsigned int *   ));

    hipMemcpy(dev_populationPosition, populationPosition, sizeof( unsigned int ***  ) , hipMemcpyHostToDevice);
    hipMemcpy(dev_map               , map               , sizeof( enum _Element *** ) , hipMemcpyHostToDevice);
    hipMemcpy(dev_simPIn            , simPIn            , sizeof( unsigned int *    ) , hipMemcpyHostToDevice);
    hipMemcpy(dev_cost              , cost              , sizeof( unsigned int **   ) , hipMemcpyHostToDevice);
    hipMemcpy(dev_simExit           , simExit           , sizeof( unsigned int *    ) , hipMemcpyHostToDevice);

    unsigned int nb_threads = 16;
    dim3 blocks((simDimX + (nb_threads-1))/nb_threads, (simDimY + (nb_threads-1))/nb_threads);
    dim3 threads(nb_threads,nb_threads);

    kernel_model1_GPU<<<blocks,threads>>>(dev_populationPosition, dev_map, dev_map, dev_simPIn, dev_cost, dev_simExit, simDimX, simDimY, simDimP);

    hipMemcpy(outMove           , dev_outMove           , sizeof( unsigned int      ) , hipMemcpyDeviceToHost);
    
    hipFree(dev_populationPosition);
    hipFree(dev_map);
    hipFree(dev_simPIn);
    hipFree(dev_cost);
    hipFree(dev_simExit);
    hipFree(dev_outMove);
}

__global__ void kernel_model1_GPU(
    unsigned int ***    dev_populationPosition,     // (*) Because change
    enum _Element ***   dev_map,                    // (*) Because change
    unsigned int *      dev_simPIn,                 // (*) Because change
    unsigned int **     dev_cost,                   // useless
    unsigned int *      dev_simExit, 
    unsigned int            simDimX, 
    unsigned int            simDimY,
    unsigned int            simDimP
){

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < simDimP){
        // position de l'individue tid
        unsigned int x = dev_populationPosition[tid][0];
        unsigned int y = dev_populationPosition[tid][1];
        // Delta à ajouté à la position pour avoir la position next step
        int deltaX =  dev_simExit[0]-x / max(dev_simExit[0]-x, dev_simExit[1]-y);
        int deltaY =  dev_simExit[1]-y / max(dev_simExit[0]-x, dev_simExit[1]-y);
        
    }
}

int cudaTest(unsigned int max) {
    int size = max;
    int a[size], b[size], c[size]; // Tableaux sur le CPU
    int *dev_a, *dev_b, *dev_c; // Tableaux sur le GPU

    // Allocation de mémoire sur le GPU
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // Initialisation des tableaux sur le CPU
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Copie des tableaux depuis le CPU vers le GPU
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Configuration du nombre de blocs et de threads par bloc
    int nb_threads = 16;
    int numBlocks = ((size + (nb_threads-1))/nb_threads, (size + (nb_threads-1))/nb_threads);
    int threadsPerBlock = nb_threads;

    // Appel du kernel CUDA
    addArrays<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_c, size);

    // Copie du résultat depuis le GPU vers le CPU
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Affichage du résultat
    for (int i = 0; i < size; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Libération de la mémoire du GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

__global__ void addArrays(int* a, int* b, int* c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}